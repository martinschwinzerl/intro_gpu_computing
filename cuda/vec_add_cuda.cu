#include <cassert>
#include <cmath>
#include <cstddef>
#include <cstdint>
#include <cstdlib>
#include <iostream>
#include <limits>
#include <random>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

/* Define the kernel function: */

__global__ void add_vec_kernel(
    double const* __restrict__ x, double const* __restrict__ y,
    double* __restrict__ z, int const n )
{
    /* blockIdx, blockDim and threadIdx are variables describing the
     * dimensions of the "grid" which are automatically provided by the
     * Cuda runtime */

    int const gid = blockIdx.x * blockDim.x + threadIdx.x;

    if( gid < n )
    {
        z[ gid ] = x[ gid ] + y[ gid ];
    }

    return;
}

int main( void )
{
    /* ---------------------------------------------------------------------- */
    /* prepare the host vectors: */

    int32_t const N = int32_t{ 10000 };

    std::vector< double > x( N, double{ 0.0 } );
    std::vector< double > y( N, double{ 0.0 } );
    std::vector< double > z( N, double{ 0.0 } );

    std::mt19937_64 prng( 20181205u );
    std::uniform_real_distribution< double >
        dist( double{ -10. }, double{ +10. } );

    for( int32_t ii = int32_t{ 0 } ; ii < N ; ++ii )
    {
        x[ ii ] = dist( prng );
        y[ ii ] = dist( prng );
    }

    hipError_t cu_err;

    /* --------------------------------------------------------------------- */
    /* use the "default" / "first" Cuda device for the program: */

    int device = int{ 0 };
    ::hipGetDevice( &device );
    cu_err = ::hipDeviceSynchronize();
    assert( cu_err == ::hipSuccess );

    /* --------------------------------------------------------------------- */
    /* Allocate the buffers on the device */
    /* x_arg, y_arg, z_arg ... handles on the host side managing buffers in *
     * the device memory */

    double* x_arg = nullptr;
    double* y_arg = nullptr;
    double* z_arg = nullptr;

    ::hipMalloc( &x_arg, sizeof( double ) * N );
    ::hipMalloc( &y_arg, sizeof( double ) * N );
    ::hipMalloc( &z_arg, sizeof( double ) * N );

    /* --------------------------------------------------------------------- */
    /* Transfer x and y from host to device */

    ::hipMemcpy( x_arg, x.data(), sizeof( double ) * N, hipMemcpyHostToDevice );
    ::hipMemcpy( y_arg, y.data(), sizeof( double ) * N, hipMemcpyHostToDevice );

    /* --------------------------------------------------------------------- */
    /* execute kernel on the device */

    int32_t const threads_per_block = int32_t{ 128 };

    int32_t const num_blocks =
        ( N + threads_per_block - int32_t{ 1 } ) / threads_per_block;

    add_vec_kernel<<< num_blocks, threads_per_block >>>( x_arg, y_arg, z_arg, N );

    cu_err = ::hipPeekAtLastError();
    assert( cu_err == ::hipSuccess );

    /* -------------------------------------------------------------------- */
    /* transfer the result from the device buffer to the host buffer */

    ::hipMemcpy( z.data(), z_arg, sizeof( double ) * N, hipMemcpyDeviceToHost );

    /* ------------------------------------------------------------------- */
    /* verify that the result is correct */

    bool success = true;
    double const EPS = std::numeric_limits< double >::epsilon();

    for( int32_t ii = int32_t{ 0 } ; ii < N ; ++ii )
    {
        if( std::fabs( ( x[ ii ] + y[ ii ] ) - z[ ii ] ) > EPS )
        {
            success = false;
            break;
        }
    }

    std::cout << "Success: " << std::boolalpha << success << std::endl;

    /* -------------------------------------------------------------------- */
    /* Clean-up */

    ::hipFree( x_arg );
    ::hipFree( y_arg );
    ::hipFree( z_arg );

    return 0;
}

/* end: */
